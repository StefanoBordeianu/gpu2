
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <time.h>

#define MAX_LENGTH 5000000

#define WARPSIZE 32
// For shuffle block dim need to have the same dimension of a warp
#define BLOCKDIM WARPSIZE

#define CHAR_PER_BIN  6
#define ALPHABET_SIZE 26
#define BIN_NUM       ((ALPHABET_SIZE - 1) / CHAR_PER_BIN + 1)
#define FIRST_CHAR    'a'

#define CHECK(call)                                                                 \
  {                                                                                 \
    const hipError_t err = call;                                                   \
    if (err != hipSuccess) {                                                       \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

#define CHECK_KERNELCALL()                                                          \
  {                                                                                 \
    const hipError_t err = hipGetLastError();                                     \
    if (err != hipSuccess) {                                                       \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

double get_time() // function to get the time of day in seconds
{
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return tv.tv_sec + tv.tv_usec * 1e-6;
}

void sequential_histogram(const char *data, unsigned int *histogram, const int length) {
  for (int i = 0; i < length; i++) {
    int alphabet_position = data[i] - FIRST_CHAR;
    if (alphabet_position >= 0 && alphabet_position < ALPHABET_SIZE) // check if we have an alphabet char
      histogram[alphabet_position / CHAR_PER_BIN]++; // we group the letters into blocks of CHAR_PER_BIN
  }
}

__global__ void
    histogram_kernel(const char *__restrict__ data, unsigned int *__restrict__ histogram, const int length) {
  const int i = threadIdx.x + blockIdx.x * blockDim.x;
  // All threads handle blockDim.x * gridDim.x
  // consecutive elements
  if (i < length) {
    const int alphabet_position = data[i] - FIRST_CHAR;
    if (alphabet_position >= 0 && alphabet_position < ALPHABET_SIZE)
      atomicAdd(&(histogram[alphabet_position / CHAR_PER_BIN]), 1);
  }
}

int main(int argc, char *argv[]) {
  if (argc < 2) {
    printf("Please provide a filename as an argument.\n");
    return 1;
  }

  const char *filename = argv[1];
  FILE *fp             = fopen(filename, "read");

  // unsigned char text[MAX_LENGTH];
  char *text = (char *) malloc(sizeof(char) * MAX_LENGTH);
  char *text_d;
  size_t len = 0;
  size_t read;
  unsigned int histogram[BIN_NUM]    = {0};
  unsigned int histogram_hw[BIN_NUM] = {0};
  unsigned int *histogram_d;

  if (fp == NULL)
    exit(EXIT_FAILURE);

  while ((read = getline(&text, &len, fp)) != -1) { printf("Retrieved line of length %ld:\n", read); }
  fclose(fp);

  sequential_histogram(text, histogram, len);

  CHECK(hipMalloc(&text_d, len * sizeof(char))); // allocate space for the input array on the GPU
  CHECK(hipMalloc(&histogram_d, BIN_NUM * sizeof(unsigned int)));             // and for the histogram
  CHECK(hipMemcpy(text_d, text, len * sizeof(char), hipMemcpyHostToDevice)); // copy input data on the gpu

  dim3 blocksPerGrid((len + BLOCKDIM - 1) / BLOCKDIM, 1, 1);
  dim3 threadsPerBlock(BLOCKDIM, 1, 1);
  histogram_kernel<<<blocksPerGrid, threadsPerBlock>>>(text_d, histogram_d, len);
  CHECK_KERNELCALL();
  CHECK(hipDeviceSynchronize());

  CHECK(hipMemcpy(histogram_hw,
                   histogram_d,
                   BIN_NUM * sizeof(unsigned int),
                   hipMemcpyDeviceToHost)); // copy data back from the gpu
  for (size_t i = 0; i < BIN_NUM; i++) {
    if (histogram[i] != histogram_hw[i]) {
      printf("Error on GPU at index: %ld\n", i);
      return 0;
    }
  }
  printf("ALL GPU OK\n");

  CHECK(hipFree(text_d));
  CHECK(hipFree(histogram_d));

  free(text);

  return 1;
}